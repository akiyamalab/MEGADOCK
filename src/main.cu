/*
 * Copyright (C) 2008-2019 Tokyo Institute of Technology
 *
 *
 * This file is part of MEGADOCK.
 * MEGADOCK is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * MEGADOCK is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with MEGADOCK.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

//============================================================================//
//
//  Software Name : MEGADOCK
//
//  Class Name : (main)
//
//  Contact address : Tokyo Institute of Technology, AKIYAMA Lab.
//
//============================================================================//

#include <string.h>
#include "cpu_time.h"
#include "control.h"

#ifdef CUFFT
#include <hip/hip_runtime_api.h>
#define VERSION "4.1.1 for GPU & "
#else
#define VERSION "4.1.1 for CPU & "
#endif

#ifdef MPI_DP
#define VTEXT "multiple nodes"
#else
#define VTEXT "single node"
#endif

#define LASTUPDATED "26 March, 2019"

//============================================================================//
#ifdef MPI_DP
int application(int argc,char *argv[])
#else
int main(int argc, char *argv[])
#endif
//============================================================================//
{
    Parallel  *_parallel;
    CPUTime   *_cputime;
    Control   *_control;

    struct timeval et1, et2;
    struct timeval et3, et4;
    int nproc2 = 0;
    int device_count_gpu = 0;

    gettimeofday(&et1,NULL);
    gettimeofday(&et3,NULL);


    cout << " MEGADOCK ver. "<< VERSION << VTEXT <<  endl;
    cout << "      megadock@bi.c.titech.ac.jp   lastupdated: " << LASTUPDATED << endl;
    cout << endl;

    _cputime = new CPUTime();
    _cputime->initialize();

#ifdef _OPENMP
    #pragma omp parallel
    {
        nproc2 = omp_get_num_threads();
        if(omp_get_thread_num() == 0) {
            cout << "# Using OpenMP parallelization: " << nproc2 << " threads." << endl;
        }
    }
    //printf("#OpenMP version %d\n", _OPENMP);
#else
    nproc2 = 1;
#endif //#ifdef _OPENMP

#ifdef CUFFT
    int nogpu_flag = 0;
    for (int num = 0; num < (argc-1); ++num) {
        if(!strcmp(argv[num], "-G")) {
            if(argv[num+1] != NULL) {
                if(atoi(argv[num+1]) == 0) {
                    nogpu_flag = 1;
                }
            }
        }
    }

    if(nogpu_flag != 1) {
        checkCudaErrors( hipGetDeviceCount(&device_count_gpu) );
        if (device_count_gpu == 0) {
            fprintf(stderr, "GPU Error: no devices supporting CUDA.\n");
            exit(-1);
        }

        hipDeviceProp_t deviceProp;
        checkCudaErrors( hipGetDeviceProperties(&deviceProp, 0));
        if (deviceProp.major < 1) {
            fprintf(stderr, "GPU Error: device does not support CUDA.\n");
            exit(-1);
        }

        hipSetDeviceFlags(hipDeviceMapHost);
        fprintf(stdout, "# Using CUDA device %d: %s\n", 0, deviceProp.name);
        hipSetDevice(0);
        //fprintf(stdout, "# Init CUDA device OK.\n");

        int cufft_version;
        hipfftGetVersion(&cufft_version);
        printf("# CUFFT version : %d\n", cufft_version);
    }
#endif

    _parallel = new Parallel(nproc2);
    _parallel->num_gpu(device_count_gpu); 

#ifdef CUFFT
    printf("# Number of available [threads / GPUs] : [%d / %d]\n",nproc2,device_count_gpu);
#endif

    gettimeofday(&et4,NULL);
    _cputime->t1_initialize += (et4.tv_sec-et3.tv_sec + (float)((et4.tv_usec-et3.tv_usec)*1e-6));

    _control = new Control(_cputime,_parallel);
    _control->initialize(argc,argv);
    _control->execute();

    delete _control;
    delete _parallel;

    _cputime->output();

    delete _cputime;

    gettimeofday(&et2,NULL);

    const float elapsed_time = (et2.tv_sec-et1.tv_sec + (float)((et2.tv_usec-et1.tv_usec)*1e-6));
    printf("\n");
    printf("Elapsed time                  = %8.2f sec.\n",elapsed_time);

    return 0;
}
