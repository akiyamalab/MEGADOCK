#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2020 Tokyo Institute of Technology
 */

//============================================================================//
//
//  Software Name : MEGADOCK
//
//  cuda_kernel.cu
//
//  Contact address : Tokyo Institute of Technology, AKIYAMA Lab.
//
//============================================================================//

#include <stdio.h>
#define   FMAX(a,b)  ( ((a)>(b) ) ? (a) : (b) )
#define   FMIN(a,b)  ( ((a)>(b) ) ? (b) : (a) )

__global__ void lig_vox_fill(int ng1
                             ,int na
                             ,float delta
                             ,float *radius2
                             ,float *xd
                             ,float *yd
                             ,float *zd
                             ,float *grid_coord
                             ,float *atom_coord_rotated
                             ,float *grid_r
    						 ,float grid_width)
{
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;
    const int ng2 = ng1 * ng1;
    //const int ng3 = ng2 * ng1;

    if(id < na) {
        //const int search_range = 2;
        const int search_range = (2.4 + grid_width -0.01) / grid_width;
        const int lc = ng1 * id;
        const int id3 = id * 3;
        const int i2 = atom_coord_rotated[id3  ] / grid_width + ng1 / 2;
        const int j2 = atom_coord_rotated[id3+1] / grid_width + ng1 / 2;
        const int k2 = atom_coord_rotated[id3+2] / grid_width + ng1 / 2;
        const int ia = FMAX(i2 - search_range, 0);
        const int ja = FMAX(j2 - search_range, 0);
        const int ka = FMAX(k2 - search_range, 0);
        const int ib = FMIN(i2 + search_range+1, ng1);
        const int jb = FMIN(j2 + search_range+1, ng1);
        const int kb = FMIN(k2 + search_range+1, ng1);

        for( int i = ia ; i < ib ; i++ ) {// grid around atom[l]
            if(xd[lc+i] > radius2[id]) continue;
            for( int j = ja ; j < jb ; j++ ) {
                const float d2 = xd[lc+i]+yd[lc+j];
                if(d2 > radius2[id]) continue;
                const int ij = ng2*i+ng1*j;
                for( int k = ka ; k < kb ; k++ ) {
                    const float d3 = d2 + zd[lc+k];
                    if( d3 < radius2[id] ) {// distance(grid-atom) < Van der Waals radius (* core)
                        grid_r[ij+k] = delta;    // grid[i] is filled up by atom[l]
                    }
                }
            }
        }
    }
    //*/
}


__global__ void lig_rotation(int na, float *theta, float *atom_coord_orig, float *mole_center_coord, float *atom_coord_rotated)
{
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;

    const float r11 = cos(theta[0])*cos(theta[2])  -  sin(theta[0])*cos(theta[1])*sin(theta[2]);
    const float r21 = sin(theta[0])*cos(theta[2])  +  cos(theta[0])*cos(theta[1])*sin(theta[2]);
    const float r31 = sin(theta[1])*sin(theta[2]);
    const float r12 = -cos(theta[0])*sin(theta[2])  -  sin(theta[0])*cos(theta[1])*cos(theta[2]);
    const float r22 = -sin(theta[0])*sin(theta[2])  +  cos(theta[0])*cos(theta[1])*cos(theta[2]);
    const float r32 = sin(theta[1])*cos(theta[2]);
    const float r13 = sin(theta[0])*sin(theta[1]);
    const float r23 = -cos(theta[0])*sin(theta[1]);
    const float r33 = cos(theta[1]);

    if(id < na) {
        const int id3 = id * 3;
        float     x, y, z;

        x = atom_coord_orig[id3  ] - mole_center_coord[0];
        y = atom_coord_orig[id3+1] - mole_center_coord[1];
        z = atom_coord_orig[id3+2] - mole_center_coord[2];
        atom_coord_rotated[id3  ] = r11 * x + r12 * y + r13 * z;
        atom_coord_rotated[id3+1] = r21 * x + r22 * y + r23 * z;
        atom_coord_rotated[id3+2] = r31 * x + r32 * y + r33 * z;
    }
}


__global__ void lig_calc_dis_atomgrid(int na, int ng1, float *xd, float *yd, float *zd, float *grid_coord, float *atom_coord_rotated)
{
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;
    const int nag = na * ng1;
    if(id < nag) {
        const int cur_atom = (id / ng1);
        const int cur_atom3 = cur_atom * 3;
        const int cur_grid = id % ng1;
        xd[id] = atom_coord_rotated[cur_atom3  ] - grid_coord[cur_grid];
        yd[id] = atom_coord_rotated[cur_atom3+1] - grid_coord[cur_grid];
        zd[id] = atom_coord_rotated[cur_atom3+2] - grid_coord[cur_grid];
        xd[id] *= xd[id];
        yd[id] *= yd[id];
        zd[id] *= zd[id];
    }
}

__global__ void lig_vox_init_grid(int ng3,float *grid_r,float *grid_i)
{
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < ng3) { //initialize
        grid_r[id]=0.0;
        grid_i[id]=0.0;
    }
}

__global__ void lig_vox_init_fft(int nf3,hipfftComplex *lig_in)
{
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < nf3) { //initialize
        lig_in[id] =  make_hipComplex( 0.0, 0.0);
        //lig_in[id].x=0.0;
        //lig_in[id].y=0.0;
    }
}

__global__ void ligand_voxel_set(int ng1
                                 ,hipfftComplex *lig_in
                                 ,float *grid_r
                                 ,float *grid_i)
{
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;
    const int ng2 = ng1 * ng1;
    const int ng3 = ng2 * ng1;
    const int nf1 = ng1 * 2;
    const int nf2 = nf1 * nf1;
    const int ng1_half = ng1 / 2;
    const float surface = 1.0;        // grid-assignment score (protein surface)

    //if(id == 0) for(int i=0;i<ng3;i++) if(grid_i[i]!=0.0) printf(" [%03d,%03d,%03d] :  %6.3f\n",i/ng2,i/ng1%ng1,i%ng1,grid_i[i]);
    //if(id == 0) for(int i=0;i<ng3;i++) printf(" [%03d,%03d,%03d] :  %6.3f\n",i/ng2,(i/ng1)%ng1,i%ng1,grid_i[i]);

    if(id < ng3) {
        const int i = id / ng2;
        const int j = (id / ng1) % ng1;
        const int k = id % ng1;
        const int idoff = (i + ng1_half) * nf2 + (j + ng1_half) * nf1 + (k + ng1_half);

        //*
        if(grid_r[id]==surface) {// this condition judges whether surface(1.0) or temporary score(-8888.0)
            lig_in[idoff] =  make_hipComplex( grid_r[id], grid_i[id]);
        } else {
            lig_in[idoff] =  make_hipComplex( 0.0, grid_i[id]);
        }
        //*
    }
}


__global__ void lig_vox_surface_cut_CtoT(int ng1, float delta, float *grid_r)
{
    // Core score to Temporary score
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;
    const float swollen_surface = -8888.0;
    const int ng2 = ng1 * ng1;
    const int ng3 = ng2 * ng1;
    if(id < ng3) {
        const int i = id / ng2;
        const int j = (id / ng1) % ng1;
        const int k = id % ng1;
        if(i==0||i==ng1-1||j==0||j==ng1-1||k==0||k==ng1-1) { // skip border
        } else {
            if(grid_r[id]==delta) {
                if(grid_r[id-1]==0 ||
                        grid_r[id+1]==0 ||
                        grid_r[id-ng1]==0 ||
                        grid_r[id+ng1]==0 ||
                        grid_r[id-ng2]==0 ||
                        grid_r[id+ng2]==0) {
                    grid_r[id]=swollen_surface; 
                }
            }
        }
    }
}

__global__ void lig_vox_elec(int ng1,int na,float grid_width,float *_Charge,float *atom_coord_rotated,float *grid_i)
{
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;
    const int ng2 = ng1 * ng1;
    const float pad = (ng1 * grid_width / 2);

    //*
    if(id < na) {
        //if(!fabs(_Charge[id]) < 0.0001) continue;
        const int id3 = id * 3;
        const int   i = (atom_coord_rotated[id3  ] + pad) / grid_width;
        const int   j = (atom_coord_rotated[id3+1] + pad) / grid_width;
        const int   k = (atom_coord_rotated[id3+2] + pad) / grid_width;

        //grid_i[i*ng2+j*ng1+k] += _Charge[id];
        //printf(" %08d-1 :  %.2f, %.2f\n",i*ng2+j*ng1+k,grid_i[i*ng2+j*ng1+k],_Charge[id]);
        atomicAdd(&grid_i[i*ng2+j*ng1+k],_Charge[id]);

        //printf(" %08d-2 :  %.2f, %.2f\n",i*ng2+j*ng1+k,grid_i[i*ng2+j*ng1+k],_Charge[id]);
    }
    //*/
}

__global__ void lig_vox_elec_serial(int ng1,int na,float grid_width,float *_Charge,float *atom_coord_rotated,float *grid_i)
{
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;
    const int ng2 = ng1 * ng1;
    const float pad = (ng1 * grid_width / 2);

    if(id==0) {
        for( int l = 0 ; l < na ; l++ ) {
            const int l3 = l*3;
            const int i =(atom_coord_rotated[l3  ] + pad) / grid_width;
            const int j =(atom_coord_rotated[l3+1] + pad) / grid_width;
            const int k =(atom_coord_rotated[l3+2] + pad) / grid_width;
            //printf(" [%5d] [x:%12.8f,y:%12.8f,z:%12.8f] [pad:%6.3f], [%3d,%3d,%3d] \n",l,atom_coord_rotated[l3  ],atom_coord_rotated[l3+1],atom_coord_rotated[l3+2],pad,i,j,k);
            //printf(" [%5d] [x:%8.0f,y:%8.0f,z:%8.0f] [pad:%6.3f], [%3d,%3d,%3d] \n",l,atom_coord_rotated[l3  ],atom_coord_rotated[l3+1],atom_coord_rotated[l3+2],pad,i,j,k);

            //if(grid_i[i*ng2+j*ng1+k]!=0)printf(" Pos : %d, current : %f, new : %f\n",i*ng2+j*ng1+k, grid_i[i*ng2+j*ng1+k], _Charge[l]);

            grid_i[i*ng2+j*ng1+k] += _Charge[l];
        }
    }
}


__device__ void lig_vox_surface_cut_TtoO(int ng3, float delta, float *grid_r)
{
    // Temporary score to Open space score
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;
    const float swollen_surface = -8888.0; // temporary score for swollen ligand surface
    if(id < ng3) {
        if(grid_r[id]==swollen_surface) { 
            grid_r[id]=0.0;
        }
    }
}

__global__ void convolution_gpu(int nf3, float *rec_r, float *rec_i, hipfftComplex *lig_out, hipfftComplex *lig_in)
{
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;

    if(id<nf3) {
        const float lig_r = lig_out[id].x;
        const float lig_i = lig_out[id].y;

        lig_in[id] =  make_hipComplex( rec_r[id]*lig_r + rec_i[id]*lig_i, rec_r[id]*lig_i - rec_i[id]*lig_r);
        //lig_in[id].x = rec_r[id]*lig_r + rec_i[id]*lig_i;
        //lig_in[id].y = rec_r[id]*lig_i - rec_i[id]*lig_r;
    }
}

__global__ void max_pos_single(int nf3, hipfftComplex *out, float *score, int *pos)
{
    //blockDim.x = nThreads
    //score[nBlocks], pos[nBlocks] (nBlocks = nf3 / nThreads)
    //sdata[nThreads]
    extern __shared__ float sdata[];
    float mscore;

    const int thr_id  = threadIdx.x;
    const int nThreads = blockDim.x;
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;

    if(id < nf3) {
        mscore = sdata[thr_id] = hipCrealf(out[id])/nf3;
        __syncthreads();    //all threads set sdata[thr_id]

        //reduction
        for(int offset = nThreads / 2; offset > 0; offset /= 2) {
            if (thr_id < offset) {
                sdata[thr_id] = FMAX(sdata[thr_id],  sdata[thr_id +  offset]);
            }
            __syncthreads();
        }

        if (mscore == sdata[0]) {//mscore specify position of max score
            score[blockIdx.x] = sdata[0];
            pos[blockIdx.x] = id;
            //printf("   BLOCK ID:%d, sdata[0]=%f, pos=%d\n",blockIdx.x,sdata[0],id);
        }
    }
}

__global__ void max_pos_multi_set(int nf3, hipfftComplex *out, float *temp_score, int *temp_index)
{
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < nf3) {
        temp_score[id] = out[id].x;
        temp_index[id] = id;
    }
}

//, std::vector<hipfftComplex> *temp_result , thrust::vector<hipfftComplex> *temp_result
//thrust::device_ptr<hipfftComplex> *temp_result hipfftComplex *temp_result,thrust::device_ptr<hipfftComplex> temp_result
__global__ void max_pos_multi(int nf3, hipfftComplex *out, float *score, int *pos,const int num_sort,const int offset)
{
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < offset) {
        if (out[id].x < out[id+offset].x) {
            out[id].x = out[id+offset].x;
            out[id].y = out[id+offset].y;
        }
        /*
        if(id==0) {
            for(int i=0; i<num_sort*2; i++) printf(" id=%d, %f %f\n",i,out[i].x/nf3,out[i].y);
            printf("\n");
        }
        //*/
    }
    //*/
}



/*
__global__ void max_pos_multi(int nf3, hipfftComplex *out, float *score, int *pos,const int num_sort, float *temp_score, int *temp_index)
{
    //blockDim.x = nThreads,
    //score[nBlocks], pos[nBlocks] (nBlocks = nf3 / nThreads)
    //sdata[nThreads]
    extern __shared__ float sdata[];
    float mscore;
    int offset;

    const int thr_id  = threadIdx.x;
    const int nThreads = blockDim.x;
    const int id  = blockIdx.x * blockDim.x + threadIdx.x;

    //*
    if(id < nf3) {
        temp_score[id]=hipCrealf(out[id])/nf3;
        temp_index[id]=id;


        /*
        __syncthreads();    //all threads set sdata[thr_id]

        //reduction
        for(offset = nThreads / 2; offset > num_sort; ) {
            offset /= 2;
            if (thr_id < offset) {
                sdata[thr_id] = FMAX(sdata[thr_id],  sdata[thr_id +  offset]);
            }
            //if(id<1)printf(" id=%d, t=%d, off=%d\n",id,num_sort,offset);
            __syncthreads();
        }
        //if(id<1)printf(" [last] id=%d, t=%d, off=%d\n",id,num_sort,offset);

        //thrust::sort(sdata,sdata+10);

        if(id < num_sort) {
            if (mscore == sdata[id]) {//mscore specify position of max score (float equality comparison... amari yokunai)
                score[blockIdx.x] = sdata[0];
                pos[blockIdx.x] = id;
                //printf("   BLOCK ID:%d, sdata[0]=%f, pos=%d\n",blockIdx.x,sdata[0],i);
            }
        }
        //*
        if(temp_score[id] >3000) printf(" id=%d, %f %d\n",id,temp_score[id],temp_index[id]);
    }
    //*
}
//*/










